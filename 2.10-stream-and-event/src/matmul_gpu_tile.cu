#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "utils.hpp"

/* 
    使用Tiling优化的matmul的函数实现
*/
__global__ void MatmulTileKernel(float *M_device, float *N_device, float *P_device, int width){
    /* 
        对于x, y多个根据block和thread进行索引
    */
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0;

    /* 对于每一个P的元素，我们只需要循环遍历width次M和N中的元素就可以了*/
    for (int k = 0; k < width; k ++){
        P_element += M_device[y * width + k] * N_device[k * width + x];
    }

    P_device[y * width + x] = P_element;
}

/*
    使用Tiling技术
    一个tile处理的就是block, 将一个矩阵分为多个小的tile，这些tile之间的执行独立，并且可以并行
*/
void MatmulTileOnDevice(float *M_host, float *N_host, float* P_host, int width, int tile_width){
    /* 设置矩阵大小 */
    int size = width * width * sizeof(float);

    /* 分配M, N在GPU上的空间*/
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc(&M_device, size));
    CUDA_CHECK(hipMalloc(&N_device, size));

    /* 分配M, N拷贝到GPU上*/
    CUDA_CHECK(hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice));

    /* 分配P在GPU上的空间*/
    float *P_device;
    CUDA_CHECK(hipMalloc((void**)&P_device, size));

    /* 调用kernel来进行matmul计算, 在这个例子中我们用的方案是：使用一个grid，一个grid里有width*width个线程 */
    dim3 dimBlock(tile_width, tile_width);
    dim3 dimGrid(width / tile_width, width / tile_width);
    MatmulTileKernel <<<dimGrid, dimBlock>>> (M_device, N_device, P_device, size);

    /* 将结果从device拷贝回host*/
    CUDA_CHECK(hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    /* 注意要在synchronization结束之后排查kernel的错误 */
    LAST_KERNEL_CHECK(); 

    /* Free */
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}

