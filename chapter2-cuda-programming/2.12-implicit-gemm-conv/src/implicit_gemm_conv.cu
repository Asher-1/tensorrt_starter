#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include <iostream>

#include "utils.hpp"

__global__ void ImplicitGEMMConvKernel(float *M_device, float *N_device, float *P_device, int width){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0;

    for (int k = 0; k < width; k ++){
        float M_element = M_device[y * width + k];
        float N_element = N_device[k * width + x];
        P_element += M_element * N_element;
    }

    P_device[y * width + x] = P_element;
}

void ImplicitGEMMConvOnDevice(float *M_host, float *N_host, float* P_host, int width, int blockSize){
    int size = width * width * sizeof(float);

    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc(&M_device, size));
    CUDA_CHECK(hipMalloc(&N_device, size));

    CUDA_CHECK(hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice));

    float *P_device;
    CUDA_CHECK(hipMalloc(&P_device, size));

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(width / blockSize, width / blockSize);
    ImplicitGEMMConvKernel <<<dimGrid, dimBlock>>> (M_device, N_device, P_device, width);

    CUDA_CHECK(hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    LAST_KERNEL_CHECK(); 

    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}

